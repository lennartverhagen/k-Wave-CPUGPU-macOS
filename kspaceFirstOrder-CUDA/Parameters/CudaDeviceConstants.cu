/**
 * @file      CudaDeviceConstants.cu
 *
 * @author    Jiri Jaros \n
 *            Faculty of Information Technology \n
 *            Brno University of Technology \n
 *            jarosjir@fit.vutbr.cz
 *
 * @brief     The implementation file for the class for storing constants residing in CUDA constant memory.
 *
 * @version   kspaceFirstOrder 3.6
 *
 * @date      17 February  2016, 10:53 (created) \n
 *            11 February  2020, 16:21 (revised)
 *
 * @copyright Copyright (C) 2016 - 2020 SC\@FIT Research Group, Brno University of Technology, Brno, CZ.
 *
 * This file is part of the C++ extension of the [k-Wave Toolbox](http://www.k-wave.org).
 *
 * k-Wave is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * k-Wave is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied
 * warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public License for
 * more details.
 *
 * You should have received a copy of the GNU Lesser General Public License along with k-Wave.
 * If not, see [http://www.gnu.org/licenses/](http://www.gnu.org/licenses/).
 */

#include <Parameters/CudaDeviceConstants.cuh>
#include <Logger/Logger.h>

//--------------------------------------------------------------------------------------------------------------------//
//---------------------------------------------------- Variables -----------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------//

/**
 * @var      cudaDeviceConstants
 * @brief    This variable holds basic simulation constants for GPU.
 * @details  This variable holds necessary simulation constants in the CUDA GPU. memory.
 *           This variable is imported as extern into other CUDA units.
 */
__constant__ CudaDeviceConstants cudaDeviceConstants;



//--------------------------------------------------------------------------------------------------------------------//
//------------------------------------------------- Public methods ---------------------------------------------------//
//--------------------------------------------------------------------------------------------------------------------//

/**
 * Copy the structure with simulation constants to the CUDA constant memory.
 */
__host__ void CudaDeviceConstants::copyToDevice()
{
  cudaCheckErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaDeviceConstants), this, sizeof(CudaDeviceConstants)));
}// end of copyToDevice
//----------------------------------------------------------------------------------------------------------------------
